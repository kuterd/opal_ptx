#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <stdexcept>


#define CHECK_CUDA(call) do { \
    hipError_t result = (call); \
    if (result != hipSuccess) { \
        const char *errStr; \
        hipDrvGetErrorName(result, &errStr); \
        if (!errStr) { \
            errStr = "Unknown error"; \
        } \
        std::string errorMsg = "CUDA error: "; \
        errorMsg += errStr; \
        throw std::runtime_error(errorMsg); \
    } \
} while (0)

class CuModuleWrapper {
public:
    CuModuleWrapper() : cuModule(nullptr) {}

    ~CuModuleWrapper() {
        if (cuModule) {
            hipModuleUnload(cuModule);
        }
    }

    void load_ptx_code(const std::string &ptx_code) {
        // Define log buffers and their sizes

        const size_t buffer_size = 32768;
        char info_log_buffer[buffer_size];

        char error_log_buffer[buffer_size];

        hipModule_t module;
        hipJitOption options[4];
        void *option_values[4];

        // Set up options for info log buffer size and buffer
        options[0] = hipJitOptionInfoLogBufferSizeBytes;
        option_values[0] = (void*)buffer_size;

        options[1] = hipJitOptionInfoLogBuffer;
        option_values[1] = (void*)info_log_buffer;

        // Set up options for error log buffer size and buffer
        options[2] = hipJitOptionErrorLogBufferSizeBytes;
        option_values[2] = (void*)buffer_size;

        options[3] = hipJitOptionErrorLogBuffer;
        option_values[3] = (void*)error_log_buffer;

        hipError_t res = hipModuleLoadDataEx(&cuModule, ptx_code.c_str(), 4, options, option_values);
        if (res != hipSuccess) {
            const char *errStr;
            hipDrvGetErrorName(res, &errStr);

            std::string errorMessage = "Failed to load PTX code! Error: ";
            errorMessage += errStr ? errStr : "Unknown error";
            errorMessage += "\nError: " + std::string(error_log_buffer) + " Info: " + std::string(info_log_buffer);

            throw std::runtime_error(errorMessage);
        }
    }

    void launch_kernel(const std::string &kernel_name,
                       pybind11::tuple grid_dim_tuple,
                       pybind11::tuple block_dim_tuple,
                       pybind11::tuple kernel_params_tuple, unsigned int shmemSize) {

        dim3 grid_dim(grid_dim_tuple[0].cast<unsigned int>(),
                      grid_dim_tuple[1].cast<unsigned int>(),
                      grid_dim_tuple[2].cast<unsigned int>());

        dim3 block_dim(block_dim_tuple[0].cast<unsigned int>(),
                       block_dim_tuple[1].cast<unsigned int>(),
                       block_dim_tuple[2].cast<unsigned int>());

        std::vector<int64_t> kernel_arg_data;
        for (const auto& item : kernel_params_tuple) {
            kernel_arg_data.push_back(item.cast<int64_t>());
        }

        void** kernel_params = new void*[kernel_params_tuple.size() + 1];
        for (size_t i = 0; i < kernel_params_tuple.size(); ++i) {
            kernel_params[i] = reinterpret_cast<void*>(&kernel_arg_data[i]);
        }
        kernel_params[kernel_params_tuple.size()] = 0;

        hipFunction_t kernel_func;
        CHECK_CUDA(hipModuleGetFunction(&kernel_func, cuModule, kernel_name.c_str()));

        CHECK_CUDA(hipModuleLaunchKernel(kernel_func, grid_dim.x, grid_dim.y, grid_dim.z,
                           block_dim.x, block_dim.y, block_dim.z,
                           0, nullptr, kernel_params, nullptr));
        CHECK_CUDA(hipCtxSynchronize());
    }

private:
    hipCtx_t cuContext;
    hipModule_t cuModule;
};

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    pybind11::class_<CuModuleWrapper>(m, "CuModuleWrapper")
        .def(pybind11::init<>())
        .def("load_ptx_code", &CuModuleWrapper::load_ptx_code, "Load PTX code into the CUDA module")
        .def("launch_kernel", &CuModuleWrapper::launch_kernel, "Launch a CUDA kernel with specified parameters");
}
